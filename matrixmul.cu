#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "kernel.cu"
#include "dev_array.h"
#include <math.h>

using namespace std;

int main()
{
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    int N = 22;
    int SIZE = N*N;
    int cycleNum = 10000;

    // Allocate memory on the host
    vector<int> h_A(SIZE);
    vector<int> h_B(SIZE);
    vector<int> h_C(SIZE);

    // Initialize matrices on the host
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = j+1;
            h_B[i*N+j] = N-i;
        }
    }

    // Allocate memory on the device
    dev_array<int> d_A(SIZE);
    dev_array<int> d_B(SIZE);
    dev_array<int> d_C(SIZE);

    double gpuOverheadTime=0;
    clock_t begin = clock();
    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);
    clock_t end = clock();
    gpuOverheadTime = gpuOverheadTime + ((double)(end - begin) / CLOCKS_PER_SEC);

    double time_spent=0;
    double tmpGpuOverheadTime=0;

    for (int i=0;i<cycleNum;i++) {
      clock_t begin = clock();
        matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
      clock_t end = clock();
      time_spent = time_spent + ((double)(end - begin) / CLOCKS_PER_SEC);

      clock_t begin = clock();
        hipDeviceSynchronize();
        d_C.get(&h_C[0], SIZE);
        hipDeviceSynchronize();
      clock_t end = clock();
      tmpGpuOverheadTime = tmpGpuOverheadTime + ((double)(end - begin) / CLOCKS_PER_SEC);
    }
    gpuOverheadTime = gpuOverheadTime + tmpGpuOverheadTime/cycleNum;
    printf("N = %d\n",N);
    //printf("Time is calculated on %i cycles\n",cycleNum);
    //printf("\n");
    //printf("Overall time on GPU: %f\n",time_spent);
    printf("Average time on GPU: %f\n",time_spent/cycleNum+gpuOverheadTime);
    printf("-------------------------------\n");


    int *cpu_C;
    cpu_C=new int[SIZE];
    time_spent=0;

    // Now do the matrix multiplication on the CPU
    int sum;
    for (int i=0;i<cycleNum;i++) {
    sum=0;
    clock_t begin = clock();

    for (int row=0; row<N; row++){
        for (int col=0; col<N; col++){
            sum = 0.f;
            for (int n=0; n<N; n++){
                sum += h_A[row*N+n]*h_B[n*N+col];
            }
            cpu_C[row*N+col] = sum;
        }
    }
    clock_t end = clock();
    time_spent = time_spent + ((double)(end - begin) / CLOCKS_PER_SEC);
   }

    //printf("Overall time on CPU: %f\n",time_spent);
    printf("Average time on CPU: %f\n",time_spent/cycleNum);
    printf("\n");
    printf("Product of the matrices:\n");

    for (int c = 0; c < N ; c++) {
      for (int d = 0; d < N; d++)
        printf("%d\t", cpu_C[c*N+d]);

      printf("\n");
    }

    for (int c = 0; c < N ; c++) {
      for (int d = 0; d < N; d++)
        printf("%d\t", h_C[c*N+d]);

      printf("\n");
    }

    double err = 0;
    // Check the result and make sure it is correct
    for (int ROW=0; ROW < N; ROW++){
        for (int COL=0; COL < N; COL++){
            err += cpu_C[ROW * N + COL] - h_C[ROW * N + COL];
        }
    }

    cout << "Error: " << err << endl;

    return 0;
}
